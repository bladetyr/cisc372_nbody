#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"

//separating compute() into cuda functions
__global__ void accelMatrix(vector3 values, vector3* accels, vector3 d_hVel, vector3 d_hPos, double d_mass){
	//make an acceleration matrix which is NUMENTITIES squared in size;
	for (int i=0;i<NUMENTITIES;i++)
		accels[i]=&values[i*NUMENTITIES];
	//first compute the pairwise accelerations.  Effect is on the first argument.
	int i = threadIdx.x;
	//int j = blockIdx.y * blockDim.y + threadIdx.y;
	int j = 0;
	int k;
	for (i;i<NUMENTITIES;i++){
		for (j;j<NUMENTITIES;j++){
			if (i==j) {
				FILL_VECTOR(accels[i][j],0,0,0);
			}
			else{
				vector3 distance;
				for (k=0;k<3;k++) distance[k]=d_hPos[i][k]-d_hPos[j][k];
				double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
				double magnitude=sqrt(magnitude_sq);
				double accelmag=-1*GRAV_CONSTANT*d_mass[j]/magnitude_sq;
				FILL_VECTOR(accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
			}
		}
	}
}

__global__ void sumMatrix(vector3 d_hVel, vector3 d_hPos, vector3 accels){
	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	int i = threadIdx.x;
	//int j = blockIdx.y * blockDim.y + threadIdx.y;
	int j = 0;
	int k;
	if(i < NUMENTITIES) {
		vector3 accel_sum={0,0,0};
		for (j;j<NUMENTITIES;j++){
			for (k=0;k<3;k++)
				accel_sum[k]+=accels[i][j][k];
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (k=0;k<3;k++){
			d_hVel[i][k]+=accel_sum[k]*INTERVAL;
			d_hPos[i][k]=d_hVel[i][k]*INTERVAL;
		}
	}
}

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){
	vector3* dValues;
	vector3** dAccels;
	double* d_mass;

	//cuda versions of values and accels
	hipMalloc((void**)&dValues, sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	hipMalloc((void**)&dAccels, sizeof(vector3)*NUMENTITIES);
	//copy those to run on GPU
	//hipMemcpy(dValues, dValues, sizeof(float)*NUMENTITIES*NUMENTITIES, hipMemcpyHostToDevice);
	//hipMemcpy(dAccels, dAccels, sizeof(float)*NUMENTITIES, hipMemcpyHostToDevice);
	//copy the global variables too
	hipMalloc((void**)&d_hVel, sizeof(double)*NUMENTITIES);
	hipMalloc((void**)&d_hPos, sizeof(double)*NUMENTITIES);
	hipMalloc((void**)&d_mass, sizeof(double)*NUMENTITIES);

	hipMemcpy(d_hVel, hVel, sizeof(double)*NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_hPos, hPos, sizeof(double)*NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_mass, mass, sizeof(double), hipMemcpyHostToDevice);

	accelMatrix<<<1,1>>>(dValues, dAccels, *d_hVel, *d_hPos, *d_mass);
	hipDeviceSynchronize();
	sumMatrix<<<1,1>>>(*d_hVel, *d_hPos, *dAccels);
	hipDeviceSynchronize();
	//free(accels);
	//free(values);
	
	hipFree(dAccels);
	hipFree(dValues);
	hipFree(d_mass);
	hipFree(d_hVel);
	hipFree(d_hPos);
}
